#include "hip/hip_runtime.h"
//#include <stdlib.h>
#include <math.h>

#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <>

#include "vector.h"
#include "config.h"

#define KERNEL_SIZE 1024

__global__ void cuda_compute_pairwise(vector3** accels, vector3* hPos, double* mass)
{
    int i = threadIdx.x;
    int j = blockIdx.x;
    int k;
    if (i < NUMENTITIES && j < NUMENTITIES)
    {
        if (i == j)
        {
            FILL_VECTOR(accels[i][j], 0, 0, 0);
        }
        else
        {
            vector3 distance;
            for (k = 0; k < 3; k++) distance[k] = hPos[i][k] - hPos[j][k];
            double magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];
            double magnitude = sqrt(magnitude_sq);
            double accelmag = -1 * GRAV_CONSTANT * mass[j] / magnitude_sq;
            FILL_VECTOR(accels[i][j], accelmag * distance[0] / magnitude, accelmag * distance[1] / magnitude, accelmag * distance[2] / magnitude);
        }
    }
}

__global__ void cuda_compute_sumup(vector3** accels, vector3* hPos, vector3* hVel)
{
    int i = blockIdx.x;
    int j;
    int k;
    if (i < NUMENTITIES)
    {
        vector3 accel_sum = { 0,0,0 };
        for (j = 0; j < NUMENTITIES; j++)
        {
            for (k = 0; k < 3; k++)
                accel_sum[k] += accels[i][j][k];
        }
        //compute the new velocity based on the acceleration and time interval
        //compute the new position based on the velocity and time interval
        for (k = 0; k < 3; k++)
        {
            hVel[i][k] += accel_sum[k] * INTERVAL;
            hPos[i][k] = hVel[i][k] * INTERVAL;
        }
    }
}

void compute()
{
    hipError_t err;

    int i;
    //vector3* values = (vector3*)malloc(sizeof(vector3) * NUMENTITIES * NUMENTITIES);
    //vector3** accels = (vector3**)malloc(sizeof(vector3*) * NUMENTITIES);
    //make an acceleration matrix which is NUMENTITIES squared in size;
    //vector3* values;
    //hipMallocManaged(&values, sizeof(vector3) * NUMENTITIES * NUMENTITIES);
    //vector3** accels;
    //hipMallocManaged(&accels, sizeof(vector3) * NUMENTITIES);

    for (i = 0; i < NUMENTITIES; i++)
        accels[i] = &values[i * NUMENTITIES];

    //first compute the pairwise accelerations.  Effect is on the first argument.
    cuda_compute_pairwise << < KERNEL_SIZE, KERNEL_SIZE >> > (accels, hPos, mass);
    err = hipGetLastError();
    assert(err == 0);
    hipDeviceSynchronize();

    //sum up the rows of our matrix to get effect on each entity, then update velocity and position.
    cuda_compute_sumup << < KERNEL_SIZE, 1 >> > (accels, hPos, hVel);
    err = hipGetLastError();
    assert(err == 0);
    hipDeviceSynchronize();

    //hipFree(accels);
    //hipFree(values);
}

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
//void compute()
//{
//    //make an acceleration matrix which is NUMENTITIES squared in size;
//    int i, j, k;
//    vector3* values = (vector3*)malloc(sizeof(vector3) * NUMENTITIES * NUMENTITIES);
//    vector3** accels = (vector3**)malloc(sizeof(vector3*) * NUMENTITIES);
//    for (i = 0; i < NUMENTITIES; i++)
//        accels[i] = &values[i * NUMENTITIES];
//    //first compute the pairwise accelerations.  Effect is on the first argument.
//    for (i = 0; i < NUMENTITIES; i++)
//    {
//        for (j = 0; j < NUMENTITIES; j++)
//        {
//            if (i == j)
//            {
//                FILL_VECTOR(accels[i][j], 0, 0, 0);
//            }
//            else
//            {
//                vector3 distance;
//                for (k = 0; k < 3; k++) distance[k] = hPos[i][k] - hPos[j][k];
//                double magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];
//                double magnitude = sqrt(magnitude_sq);
//                double accelmag = -1 * GRAV_CONSTANT * mass[j] / magnitude_sq;
//                FILL_VECTOR(accels[i][j], accelmag * distance[0] / magnitude, accelmag * distance[1] / magnitude, accelmag * distance[2] / magnitude);
//            }
//        }
//    }
//    //sum up the rows of our matrix to get effect on each entity, then update velocity and position.
//    for (i = 0; i < NUMENTITIES; i++)
//    {
//        vector3 accel_sum = { 0,0,0 };
//        for (j = 0; j < NUMENTITIES; j++)
//        {
//            for (k = 0; k < 3; k++)
//                accel_sum[k] += accels[i][j][k];
//        }
//        //compute the new velocity based on the acceleration and time interval
//        //compute the new position based on the velocity and time interval
//        for (k = 0; k < 3; k++)
//        {
//            hVel[i][k] += accel_sum[k] * INTERVAL;
//            hPos[i][k] = hVel[i][k] * INTERVAL;
//        }
//    }
//    free(accels);
//    free(values);
//}
